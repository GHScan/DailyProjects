#include "hip/hip_runtime.h"
#include <cstdio>
#include <memory>
#include <algorithm>
#include <chrono>
#include <vector>
#include <numeric>
#include <iostream>

#include ""


#include "CUDAUtils.h"



template <typename TFunc>
inline double Timing(TFunc const &func, int times = 3) {
    using namespace std::chrono;

    if (times > 1)
        func();

    auto t = std::numeric_limits<double>::max();
    for (auto i = 0; i < times; ++i) {
        auto start = steady_clock::now();
        func();
        auto end = steady_clock::now();

        t = std::min(t, duration<double>(end - start).count());
    }

    return t;
}


inline bool IsPowerOf2(size_t i) {
    return i > 0 && ((i - 1) & i) == 0;
}


#define USE(v)  do { volatile auto _v2 = v; } while(0)


template<typename T>
__device__ inline void Swap(T &a, T& b) {
    auto c = a; 
    a = b;
    b = c;
}


#define MAX_CUDA_THREADS  1024


struct Bitonic_Naive {

    template<typename TIter>
    static void Kernel(TIter first, TIter mid, bool ascent) {
        for (auto p = first, q = mid; p != mid; ++p, ++q) {
            if ((*p > *q) == ascent)
                std::iter_swap(p, q);
        }
    }


    template<typename TIter>
    static void Bitonic_Sort(TIter first, TIter mid, TIter last, bool ascent) {
        if (first + 1 == last) return;

        Kernel(first, mid, ascent);
        Bitonic_Sort(first, (mid - first) / 2 + first, mid, ascent);
        Bitonic_Sort(mid, (last - mid) / 2 + mid, last, ascent);
    }


    template<typename TIter>
    static void Sort(TIter first, TIter last, bool ascent) {
        auto size = last - first;
        assert(IsPowerOf2(size));
        if (size == 1) return;

        auto mid = size / 2 + first;
        
        Sort(first, mid, true);
        Sort(mid, last, false);
        Bitonic_Sort(first, mid, last, ascent);
    }
};


struct Bitonic_Unroll1 {

    template<typename TIter>
    static void Kernel(TIter first, TIter last, size_t span, bool ascent) {
        for (auto p = first; p != last; p += span) {
            for (auto q = p + span; p != q; ++p) {
                if ((p[0] > p[span]) == ascent)
                    std::iter_swap(p, p + span);
            }
        }
    }


    template<typename TIter>
    static void Bitonic_Sort(TIter first, TIter mid, TIter last, bool ascent) {
        for (auto span = mid - first; span >= 1; span /= 2) {
            Kernel(first, last, span, ascent);
        }
    }


    template<typename TIter>
    static void Sort(TIter first, TIter last, bool ascent) {
        auto size = last - first;
        assert(IsPowerOf2(size));
        if (size == 1) return;

        auto mid = size / 2 + first;

        Sort(first, mid, true);
        Sort(mid, last, false);
        Bitonic_Sort(first, mid, last, ascent);
    }
};


struct Bitonic_Unroll2 {

    template<typename T>
    static void Sort(T *ptr, size_t size, bool ascent) {
        assert(IsPowerOf2(size));

        for (auto dirSpan = 2; dirSpan <= size; dirSpan *= 2) {
            for (auto span = size / 2; span >= 1; span /= 2) {
                for (size_t i = 0; i < size; ++i) {
                    if ((i / span) % 2 == 0) {
                        auto dir = (i / dirSpan) % 2 == 0 ? ascent : !ascent;
                        if ((ptr[i] > ptr[i + span]) == dir) {
                            std::iter_swap(ptr + i, ptr + i + span);
                        }
                    }
                }
            }
        }
    }
};


template<typename T>
__global__ void Bitonic_GPU1_Kernel(T *ptr, size_t size, bool ascent) {
    auto tid = threadIdx.x;
    auto elemCountPerThread = size / blockDim.x;
    auto first = tid * elemCountPerThread;
    auto last = first + elemCountPerThread;

    __shared__ T localPtr[4096];
    for (auto i = first; i != last; ++i)
        localPtr[i] = ptr[i];
    __syncthreads();
        

    for (auto dirSpan = 2; dirSpan <= size; dirSpan *= 2) {
        for (auto span = size / 2; span >= 1; span /= 2) {
#pragma unroll
            for (auto i = first; i != last; ++i) {
                if ((i / span) % 2 == 0) {
                    auto dir = (i / dirSpan) % 2 == 0 ? ascent : !ascent;
                    if ((localPtr[i] > localPtr[i + span]) == dir) {
                        Swap(localPtr[i], localPtr[i + span]);
                    }
                }
            }

            __syncthreads();
        }
    }

    for (auto i = first; i != last; ++i)
        ptr[i] = localPtr[i];
}

struct Bitonic_GPU1 {

    template<typename T>
    static void Sort(CUDAArrayPtr<T> devPtr, bool ascent, size_t maxThread) {

        assert(IsPowerOf2(devPtr->Length));

        auto threadCount = std::min(maxThread, devPtr->Length);
        Bitonic_GPU1_Kernel << < 1, threadCount >> > (devPtr->Ptr, devPtr->Length, ascent);

        devPtr->Device->CheckLastError();
    }


    template<typename T>
    static void Sort(
        CUDADevicePtr device, 
        T *ptr, size_t size, bool ascent, size_t maxThread = 512) {

        auto devPtr = device->Alloc<T>(size);
        device->Copy(devPtr, ptr);
        Sort(devPtr, ascent, maxThread);
        device->Copy(ptr, devPtr);
    }
};


template<bool ascent, typename T>
__global__ void Bitonic_GPU2_Kernel1(T *ptr, int dirSpan, int span) {
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if ((i & span) == 0) {
        auto dir = (i & dirSpan) == 0 ? ascent : !ascent;
        if ((ptr[i] > ptr[i + span]) == dir) {
            Swap(ptr[i], ptr[i + span]);
        }
    }
}

template<bool ascent, typename T>
__global__ void Bitonic_GPU2_Kernel2(T *ptr, int dirSpan, int initSpan) {

    auto i = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ T localPtr[MAX_CUDA_THREADS];
    localPtr[threadIdx.x] = ptr[i];
    __syncthreads();

    auto dir = (i & dirSpan) == 0 ? ascent : !ascent;
#pragma unroll
    for (auto span = initSpan; span >= 1; span >>= 1) {
        if ((i & span) == 0) {
            if ((localPtr[threadIdx.x] > localPtr[threadIdx.x + span]) == dir) {
                Swap(localPtr[threadIdx.x], localPtr[threadIdx.x + span]);
            }
        }

        __syncthreads();
    }

    ptr[i] = localPtr[threadIdx.x];
}

struct Bitonic_GPU2 {

    template<bool ascent, typename T>
    static void Sort(CUDAArrayPtr<T> devPtr) {
        assert(IsPowerOf2(devPtr->Length));

        auto size = int(devPtr->Length);
        for (auto dirSpan = 2; dirSpan <= size; dirSpan <<= 1) {
            auto span = size >> 1;

            for (; span >= MAX_CUDA_THREADS; span >>= 1) {

                Bitonic_GPU2_Kernel1 
                    <ascent, T> 
                    << < devPtr->Length / MAX_CUDA_THREADS, MAX_CUDA_THREADS >> > 
                    (devPtr->Ptr, dirSpan, span);

                devPtr->Device->CheckLastError();
            }

            {
                auto threadCount = std::min(size, MAX_CUDA_THREADS);
                auto blockCount = size / threadCount;

                Bitonic_GPU2_Kernel2 
                    <ascent, T> 
                    << <blockCount, threadCount >> >
                    (devPtr->Ptr, dirSpan, span);

                devPtr->Device->CheckLastError();
            }
        }
    }


    template<bool ascent, typename T>
    static void Sort(CUDADevicePtr device, T *ptr, size_t size) {
        auto devPtr = device->Alloc<T>(size);
        device->Copy(devPtr, ptr);
        Sort<ascent>(devPtr);
        device->Copy(ptr, devPtr);
    }
};



static void Test(CUDADevicePtr device) {
    for (auto i = 0; i < 12; ++i) {
        std::vector<int> range(1 << i);
        iota(range.begin(), range.end(), 0);
        auto temp(range);

        random_shuffle(temp.begin(), temp.end());
        Bitonic_Naive::Sort(temp.begin(), temp.end(), true);
        assert(std::equal(temp.begin(), temp.end(), range.begin()));

        random_shuffle(temp.begin(), temp.end());
        Bitonic_Unroll1::Sort(temp.begin(), temp.end(), true);
        assert(std::equal(temp.begin(), temp.end(), range.begin()));

        random_shuffle(temp.begin(), temp.end());
        Bitonic_Unroll2::Sort(&temp[0], temp.size(), true);
        assert(std::equal(temp.begin(), temp.end(), range.begin()));

        random_shuffle(temp.begin(), temp.end());
        Bitonic_GPU1::Sort(device, &temp[0], temp.size(), true);
        assert(std::equal(temp.begin(), temp.end(), range.begin()));

        random_shuffle(temp.begin(), temp.end());
        Bitonic_GPU2::Sort<true>(device, &temp[0], temp.size());
        assert(std::equal(temp.begin(), temp.end(), range.begin()));
    }
}


static void Benchmark(CUDADevicePtr device) {
    constexpr size_t kSize = 1 << 12;
    constexpr size_t kLoop = 100;

    std::vector<int> shuffled(kSize);
    iota(shuffled.begin(), shuffled.end(), 0);
    random_shuffle(shuffled.begin(), shuffled.end());
    {
        auto temp(shuffled);

        auto baseline = Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                temp.assign(shuffled.begin(), shuffled.end());
                USE(temp.front());
            }
        }) / kLoop;


        printf("%-24s=%f\n", "std::sort", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                temp.assign(shuffled.begin(), shuffled.end());
                sort(temp.begin(), temp.end());
                USE(temp.front());
            }
        }) / kLoop - baseline);

        printf("%-24s=%f\n", "bitonic_naive", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                temp.assign(shuffled.begin(), shuffled.end());
                Bitonic_Naive::Sort(temp.begin(), temp.end(), true);
                USE(temp.front());
            }
        }) / kLoop - baseline);

        printf("%-24s=%f\n", "bitonic_unroll1", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                temp.assign(shuffled.begin(), shuffled.end());
                Bitonic_Unroll1::Sort(temp.begin(), temp.end(), true);
                USE(temp.front());
            }
        }) / kLoop - baseline);

        printf("%-24s=%f\n", "bitonic_unroll2", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                temp.assign(shuffled.begin(), shuffled.end());
                Bitonic_Unroll2::Sort(&temp[0], temp.size(), true);
                USE(temp.front());
            }
        }) / kLoop - baseline);
    }
    {
        auto devShuffled = device->Alloc<int>(kSize);
        device->Copy(devShuffled, &shuffled[0]);
        auto devTemp = device->Alloc<int>(kSize);

        auto baseline = Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                device->Copy(devTemp, devShuffled);
            }
            device->Synchronize();
        }) / kLoop;

        printf("%-24s=%f\n", "bitonic_gpu1,128", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                device->Copy(devTemp, devShuffled);
                Bitonic_GPU1::Sort(devTemp, true, 128);
            }
            device->Synchronize();
        }) / kLoop - baseline);

        printf("%-24s=%f\n", "bitonic_gpu1,1024", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                device->Copy(devTemp, devShuffled);
                Bitonic_GPU1::Sort(devTemp, true, 1024);
            }
            device->Synchronize();
        }) / kLoop - baseline);

        printf("%-24s=%f\n", "bitonic_gpu2", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                device->Copy(devTemp, devShuffled);
                Bitonic_GPU2::Sort<true>(devTemp);
            }
            device->Synchronize();
        }) / kLoop - baseline);
    }
}


int main() {
    try {
        auto device = std::make_shared<CUDADevice>();

        Test(device);
#ifdef NDEBUG
        Benchmark(device);
#endif
    } catch(std::exception const &e) {
        std::cerr << e.what() << std::endl;
    }


    return 0;
}
