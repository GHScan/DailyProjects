#include "hip/hip_runtime.h"
#include <cstdio>
#include <memory>
#include <algorithm>
#include <chrono>
#include <vector>
#include <numeric>
#include <iostream>

#include ""


#include "CUDAUtils.h"



template <typename TFunc>
inline double Timing(TFunc &&func, int times = 3) {
    using namespace std::chrono;

    if (times > 1)
        func();

    auto t = std::numeric_limits<double>::max();
    for (auto i = 0; i < times; ++i) {
        auto start = high_resolution_clock::now();
        func();
        auto end = high_resolution_clock::now();
        t = std::min(t, duration<double>(end - start).count());
    }

    return t;
}


inline bool IsPowerOf2(size_t i) {
    return i > 0 && ((i - 1) & i) == 0;
}


#define USE(v)  do { volatile auto _v2 = v; } while(0)


struct Bitonic_v0 {
    template<typename TIter>
    static void Kernel(TIter first, TIter mid, bool ascent) {
        for (auto p = first, q = mid; p != mid; ++p, ++q) {
            if ((*p > *q) == ascent)
                std::iter_swap(p, q);
        }
    }


    template<typename TIter>
    static void Bitonic_Sort(TIter first, TIter mid, TIter last, bool ascent) {
        if (first + 1 == last) return;

        Kernel(first, mid, ascent);
        Bitonic_Sort(first, (mid - first) / 2 + first, mid, ascent);
        Bitonic_Sort(mid, (last - mid) / 2 + mid, last, ascent);
    }


    template<typename TIter>
    static void Sort(TIter first, TIter last, bool ascent) {
        auto size = last - first;
        assert(IsPowerOf2(size));
        if (size == 1) return;
        auto mid = size / 2 + first;
        
        Sort(first, mid, true);
        Sort(mid, last, false);
        Bitonic_Sort(first, mid, last, ascent);
    }
};


struct Bitonic_v1 {
    template<typename TIter>
    static void Kernel(TIter first, TIter last, size_t span, bool ascent) {
        for (auto p = first; p != last; p += span) {
            for (auto q = p + span; p != q; ++p) {
                if ((p[0] > p[span]) == ascent)
                    std::iter_swap(p, p + span);
            }
        }
    }


    template<typename TIter>
    static void Bitonic_Sort(TIter first, TIter mid, TIter last, bool ascent) {
        for (auto span = mid - first; span >= 1; span /= 2) {
            Kernel(first, last, span, ascent);
        }
    }


    template<typename TIter>
    static void Sort(TIter first, TIter last, bool ascent) {
        auto size = last - first;
        assert(IsPowerOf2(size));
        if (size == 1) return;
        auto mid = size / 2 + first;

        Sort(first, mid, true);
        Sort(mid, last, false);
        Bitonic_Sort(first, mid, last, ascent);
    }
};


struct Bitonic_v2 {
    template<typename T>
    static void Sort(T *ptr, size_t size, bool ascent) {
        assert(IsPowerOf2(size));

        for (auto dirSpan = 2; dirSpan <= size; dirSpan *= 2) {
            for (auto span = size / 2; span >= 1; span /= 2) {
                for (size_t i = 0; i < size; ++i) {
                    if ((i / span) % 2 == 0) {
                        if ((ptr[i] > ptr[i + span]) == ((i / dirSpan) % 2 == 0 ? ascent : !ascent)) {
                            auto v = ptr[i]; ptr[i] = ptr[i + span]; ptr[i + span] = v;
                        }
                    }
                }
            }
        }
    }
};


template<typename T>
__global__ void Bitonic_v3_Kernel(T *ptr, size_t size, bool ascent) {
    auto tid = threadIdx.x;
    auto elemCountPerThread = size / blockDim.x;
    auto first = tid * elemCountPerThread;
    auto last = first + elemCountPerThread;

    __shared__ T localPtr[4096];
    for (auto i = first; i != last; ++i)
        localPtr[i] = ptr[i];
    __syncthreads();
        

    for (auto dirSpan = 2; dirSpan <= size; dirSpan *= 2) {
        for (auto span = size / 2; span >= 1; span /= 2) {
#pragma unroll
            for (auto i = first; i != last; ++i) {
                if ((i / span) % 2 == 0) {
                    if ((localPtr[i] > localPtr[i + span]) == ((i / dirSpan) % 2 == 0 ? ascent : !ascent)) {
                        auto v = localPtr[i]; localPtr[i] = localPtr[i + span]; localPtr[i + span] = v;
                    }
                }
            }

            __syncthreads();
        }
    }

    for (auto i = first; i != last; ++i)
        ptr[i] = localPtr[i];
}

struct Bitonic_v3 {
    template<typename T>
    static void Sort(std::shared_ptr<CUDAArray<T>> devPtr, bool ascent, size_t maxThread) {
        assert(IsPowerOf2(devPtr->Length));

        auto threadCount = std::min(maxThread, devPtr->Length);
        Bitonic_v3_Kernel << < 1, threadCount >> > (devPtr->Ptr, devPtr->Length, ascent);

        devPtr->Device->CheckLastError();
    }


    template<typename T>
    static void Sort(std::shared_ptr<CUDADevice> device, T *ptr, size_t size, bool ascent, size_t maxThread = 512) {
        auto devPtr = device->Alloc<T>(size);
        device->Copy(devPtr, ptr);
        Sort(devPtr, ascent, maxThread);
        device->Copy(ptr, devPtr);
    }
};


template<typename T>
__global__ void Bitonic_v4_Kernel(T *ptr, size_t size, size_t dirSpan, size_t span, bool ascent) {
    auto totalThreadCount = gridDim.x * blockDim.x;
    auto elemCountPerThread = totalThreadCount < size ? size / totalThreadCount : 1;
    auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    auto first = tid * elemCountPerThread;
    auto last = first + elemCountPerThread;

    for (auto i = first; i != last; ++i) {
        if (i < size && (i / span) % 2 == 0) {
            if ((ptr[i] > ptr[i + span]) == ((i / dirSpan) % 2 == 0 ? ascent : !ascent)) {
                auto v = ptr[i]; ptr[i] = ptr[i + span]; ptr[i + span] = v;
            }
        }
    }
}

struct Bitonic_v4 {
    template<typename T>
    static void Sort(std::shared_ptr<CUDAArray<T>> devPtr, bool ascent) {
        assert(IsPowerOf2(devPtr->Length));

        size_t threadCount = 1024;
        size_t blockCount = std::max<size_t>(devPtr->Length / threadCount, 1);
        for (auto dirSpan = 2; dirSpan <= devPtr->Length; dirSpan *= 2) {
            for (auto span = devPtr->Length / 2; span >= 1; span /= 2) {
                Bitonic_v4_Kernel << < blockCount, threadCount >> > (devPtr->Ptr, devPtr->Length, dirSpan, span, ascent);
            }
        }

        devPtr->Device->CheckLastError();
    }


    template<typename T>
    static void Sort(std::shared_ptr<CUDADevice> device, T *ptr, size_t size, bool ascent) {
        auto devPtr = device->Alloc<T>(size);
        device->Copy(devPtr, ptr);
        Sort(devPtr, ascent);
        device->Copy(ptr, devPtr);
    }
};


static void Test(std::shared_ptr<CUDADevice> device) {

    for (auto i = 0; i < 12; ++i) {
        std::vector<int> range(1 << i);
        iota(range.begin(), range.end(), 0);
        auto temp(range);

        random_shuffle(temp.begin(), temp.end());
        Bitonic_v0::Sort(temp.begin(), temp.end(), true);
        assert(std::equal(temp.begin(), temp.end(), range.begin()));

        random_shuffle(temp.begin(), temp.end());
        Bitonic_v1::Sort(temp.begin(), temp.end(), true);
        assert(std::equal(temp.begin(), temp.end(), range.begin()));

        random_shuffle(temp.begin(), temp.end());
        Bitonic_v2::Sort(&temp[0], temp.size(), true);
        assert(std::equal(temp.begin(), temp.end(), range.begin()));

        random_shuffle(temp.begin(), temp.end());
        Bitonic_v3::Sort(device, &temp[0], temp.size(), true);
        assert(std::equal(temp.begin(), temp.end(), range.begin()));

        random_shuffle(temp.begin(), temp.end());
        Bitonic_v4::Sort(device, &temp[0], temp.size(), true);
        assert(std::equal(temp.begin(), temp.end(), range.begin()));
    }
}


static void Benchmark(std::shared_ptr<CUDADevice> device) {
    constexpr size_t kSize = 1 << 12;
    constexpr size_t kLoop = 100;

    std::vector<int> shuffled(kSize);
    iota(shuffled.begin(), shuffled.end(), 0);
    random_shuffle(shuffled.begin(), shuffled.end());
    {
        auto temp(shuffled);

        auto baseline = Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                temp.assign(shuffled.begin(), shuffled.end());
                USE(temp.front());
            }
        }) / kLoop;


        printf("%-24s=%f\n", "qsort", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                temp.assign(shuffled.begin(), shuffled.end());
                sort(temp.begin(), temp.end());
                USE(temp.front());
            }
        }) / kLoop - baseline);

        printf("%-24s=%f\n", "bitonic_0", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                temp.assign(shuffled.begin(), shuffled.end());
                Bitonic_v0::Sort(temp.begin(), temp.end(), true);
                USE(temp.front());
            }
        }) / kLoop - baseline);

        printf("%-24s=%f\n", "bitonic_1", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                temp.assign(shuffled.begin(), shuffled.end());
                Bitonic_v1::Sort(temp.begin(), temp.end(), true);
                USE(temp.front());
            }
        }) / kLoop - baseline);

        printf("%-24s=%f\n", "bitonic_2", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                temp.assign(shuffled.begin(), shuffled.end());
                Bitonic_v2::Sort(&temp[0], temp.size(), true);
                USE(temp.front());
            }
        }) / kLoop - baseline);
    }
    {
        auto devShuffled = device->Alloc<int>(kSize);
        device->Copy(devShuffled, &shuffled[0]);
        auto devTemp = device->Alloc<int>(kSize);

        auto baseline = Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                device->Copy(devTemp, devShuffled);
            }
            device->Synchronize();
        }) / kLoop;

        printf("%-24s=%f\n", "bitonic_3,128", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                device->Copy(devTemp, devShuffled);
                Bitonic_v3::Sort(devTemp, true, 128);
            }
            device->Synchronize();
        }) / kLoop - baseline);

        printf("%-24s=%f\n", "bitonic_3,1024", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                device->Copy(devTemp, devShuffled);
                Bitonic_v3::Sort(devTemp, true, 1024);
            }
            device->Synchronize();
        }) / kLoop - baseline);

        printf("%-24s=%f\n", "bitonic_4", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                device->Copy(devTemp, devShuffled);
                Bitonic_v4::Sort(devTemp, true);
            }
            device->Synchronize();
        }) / kLoop - baseline);
    }
}


int main() {
    try {
        auto device = std::make_shared<CUDADevice>();

        Test(device);
#ifdef NDEBUG
        Benchmark(device);
#endif
    } catch(std::exception const &e) {
        std::cerr << e.what() << std::endl;
    }


    return 0;
}