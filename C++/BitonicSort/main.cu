#include "hip/hip_runtime.h"
#include <cstdio>
#include <memory>
#include <algorithm>
#include <chrono>
#include <vector>
#include <numeric>
#include <iostream>

#include ""


#include "CUDAUtils.h"



template <typename TFunc>
inline double Timing(TFunc &&func, int times = 3) {
    using namespace std::chrono;

    if (times > 1)
        func();

    auto t = std::numeric_limits<double>::max();
    for (auto i = 0; i < times; ++i) {
        auto start = high_resolution_clock::now();
        func();
        auto end = high_resolution_clock::now();
        t = std::min(t, duration<double>(end - start).count());
    }

    return t;
}


inline bool IsPowerOf2(size_t n) {
    return n > 0 && ((n - 1) & n) == 0;
}


#define USE(v)  do { volatile auto _v2 = v; } while(0)


struct Bitonic_v0 {
    template<typename TIter>
    static void Kernel(TIter first, TIter mid, bool ascent) {
        for (auto p = first, q = mid; p != mid; ++p, ++q) {
            if ((*p > *q) == ascent)
                std::iter_swap(p, q);
        }
    }


    template<typename TIter>
    static void Bitonic_Sort(TIter first, TIter mid, TIter last, bool ascent) {
        if (first + 1 == last) return;

        Kernel(first, mid, ascent);
        Bitonic_Sort(first, (mid - first) / 2 + first, mid, ascent);
        Bitonic_Sort(mid, (last - mid) / 2 + mid, last, ascent);
    }


    template<typename TIter>
    static void Sort(TIter first, TIter last, bool ascent) {
        auto size = last - first;
        assert(IsPowerOf2(size));
        if (size == 1) return;
        auto mid = size / 2 + first;
        
        Sort(first, mid, true);
        Sort(mid, last, false);
        Bitonic_Sort(first, mid, last, ascent);
    }
};


struct Bitonic_v1 {
    template<typename TIter>
    static void Kernel(TIter first, TIter last, size_t span, bool ascent) {
        for (auto p = first; p != last; p += span) {
            for (auto q = p + span; p != q; ++p) {
                if ((p[0] > p[span]) == ascent)
                    std::iter_swap(p, p + span);
            }
        }
    }


    template<typename TIter>
    static void Bitonic_Sort(TIter first, TIter mid, TIter last, bool ascent) {
        for (auto span = mid - first; span >= 1; span /= 2) {
            Kernel(first, last, span, ascent);
        }
    }


    template<typename TIter>
    static void Sort(TIter first, TIter last, bool ascent) {
        auto size = last - first;
        assert(IsPowerOf2(size));
        if (size == 1) return;
        auto mid = size / 2 + first;

        Sort(first, mid, true);
        Sort(mid, last, false);
        Bitonic_Sort(first, mid, last, ascent);
    }
};


struct Bitonic_v2 {
    template<typename T>
    static void Sort(T *p, size_t size, bool ascent) {
        assert(IsPowerOf2(size));

        for (auto dirSpan = 2; dirSpan <= size; dirSpan *= 2) {
            for (auto span = size / 2; span >= 1; span /= 2) {
                for (size_t i = 0; i < size; ++i) {
                    if ((i / span) % 2 == 0) {
                        if ((p[i] > p[i + span]) == ((i / dirSpan) % 2 == 0 ? ascent : !ascent)) {
                            auto v = p[i]; p[i] = p[i + span]; p[i + span] = v;
                        }
                    }
                }
            }
        }
    }
};


template<typename T>
__global__ void Bitonic_v3_Kernel(T *ptr, size_t size, bool ascent) {
    auto tid = threadIdx.x;
    auto win = size / blockDim.x;
    auto first = tid * win;
    auto last = first + win;

    __shared__ T p[4096];
    for (auto i = first; i != last; ++i)
        p[i] = ptr[i];
    __syncthreads();
        

    for (auto dirSpan = 2; dirSpan <= size; dirSpan *= 2) {
        for (auto span = size / 2; span >= 1; span /= 2) {
#pragma unroll
            for (auto i = first; i != last; ++i) {
                if ((i / span) % 2 == 0) {
                    if ((p[i] > p[i + span]) == ((i / dirSpan) % 2 == 0 ? ascent : !ascent)) {
                        auto v = p[i]; p[i] = p[i + span]; p[i + span] = v;
                    }
                }
            }

            __syncthreads();
        }
    }

    for (auto i = first; i != last; ++i)
        ptr[i] = p[i];
}

struct Bitonic_v3 {
    template<typename T>
    static void Sort(std::shared_ptr<CUDAArray<T>> a, bool ascent, size_t maxThread) {
        assert(IsPowerOf2(a->Length));

        auto threadCount = std::min(maxThread, a->Length);
        Bitonic_v3_Kernel << < 1, threadCount >> > (a->Ptr, a->Length, ascent);

        a->Device->CheckLastError();
    }


    template<typename T>
    static void Sort(std::shared_ptr<CUDADevice> device, T *a, size_t size, bool ascent, size_t maxThread = 512) {
        auto da = device->Alloc<T>(size);
        device->Copy(da, a);
        Sort(da, ascent, maxThread);
        device->Copy(a, da);
    }
};


static void Test(std::shared_ptr<CUDADevice> device) {

    for (auto i = 0; i < 12; ++i) {
        std::vector<int> range(1 << i);
        iota(range.begin(), range.end(), 0);
        auto a(range);

        random_shuffle(a.begin(), a.end());
        Bitonic_v0::Sort(a.begin(), a.end(), true);
        assert(std::equal(a.begin(), a.end(), range.begin()));

        random_shuffle(a.begin(), a.end());
        Bitonic_v1::Sort(a.begin(), a.end(), true);
        assert(std::equal(a.begin(), a.end(), range.begin()));

        random_shuffle(a.begin(), a.end());
        Bitonic_v2::Sort(&a[0], a.size(), true);
        assert(std::equal(a.begin(), a.end(), range.begin()));

        random_shuffle(a.begin(), a.end());
        Bitonic_v3::Sort(device, &a[0], a.size(), true);
        assert(std::equal(a.begin(), a.end(), range.begin()));
    }
}


static void Benchmark(std::shared_ptr<CUDADevice> device) {
    constexpr size_t kSize = 1 << 12;
    constexpr size_t kLoop = 100;

    std::vector<int> a(kSize);
    iota(a.begin(), a.end(), 0);
    random_shuffle(a.begin(), a.end());
    {
        auto b(a);

        auto baseline = Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                b.assign(a.begin(), a.end());
                USE(b.front());
            }
        }) / kLoop;


        printf("%-24s=%f\n", "qsort", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                b.assign(a.begin(), a.end());
                sort(b.begin(), b.end());
                USE(b.front());
            }
        }) / kLoop - baseline);

        printf("%-24s=%f\n", "bitonic_0", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                b.assign(a.begin(), a.end());
                Bitonic_v0::Sort(b.begin(), b.end(), true);
                USE(b.front());
            }
        }) / kLoop - baseline);

        printf("%-24s=%f\n", "bitonic_1", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                b.assign(a.begin(), a.end());
                Bitonic_v1::Sort(b.begin(), b.end(), true);
                USE(b.front());
            }
        }) / kLoop - baseline);

        printf("%-24s=%f\n", "bitonic_2", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                b.assign(a.begin(), a.end());
                Bitonic_v2::Sort(&b[0], b.size(), true);
                USE(b.front());
            }
        }) / kLoop - baseline);
    }
    {
        auto da = device->Alloc<int>(kSize);
        device->Copy(da, &a[0]);
        auto db = device->Alloc<int>(kSize);

        auto baseline = Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                device->Copy(db, da);
            }
            device->Synchronize();
        }) / kLoop;

        printf("%-24s=%f\n", "bitonic_3,128", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                device->Copy(db, da);
                Bitonic_v3::Sort(db, true, 128);
            }
            device->Synchronize();
        }) / kLoop - baseline);

        printf("%-24s=%f\n", "bitonic_3,256", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                device->Copy(db, da);
                Bitonic_v3::Sort(db, true, 256);
            }
            device->Synchronize();
        }) / kLoop - baseline);

        printf("%-24s=%f\n", "bitonic_3,512", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                device->Copy(db, da);
                Bitonic_v3::Sort(db, true, 512);
            }
            device->Synchronize();
        }) / kLoop - baseline);

        printf("%-24s=%f\n", "bitonic_3,1024", Timing([&]() {
            for (size_t i = 0; i < kLoop; ++i) {
                device->Copy(db, da);
                Bitonic_v3::Sort(db, true, 1024);
            }
            device->Synchronize();
        }) / kLoop - baseline);
    }
}


int main() {
    try {
        auto device = std::make_shared<CUDADevice>();

        Test(device);
#ifdef NDEBUG
        Benchmark(device);
#endif
    } catch(std::exception const &e) {
        std::cerr << e.what() << std::endl;
    }


    return 0;
}