#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#include <vector>
#include <memory>

#include "hip/hip_runtime.h"
#include ""
#include "FractalRenderer.h"

#if USE_DOUBLE
#define ufmod   fmod
#else
#define ufmod   fmodf
#endif

static const int kBlockSize = 32;

#define cudaCheck(call) { cudaCheckError((call), __FILE__, __LINE__); }
inline void cudaCheckError(hipError_t error, const char *file, int line)
{
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s %s %d\n", hipGetErrorString(error), file, line);
        throw std::exception(hipGetErrorString(error));
    }
}

__device__ static int HSV2RGB(TFloat H, TFloat S, TFloat V)
{
    TFloat C = V * S;
    TFloat H1 = H * 6;
    TFloat X = C * (1 - fabs(ufmod(H1, 2.0) - 1));
    TFloat R1, G1, B1;
    switch ((int)floor(H1))
    {
    case 0:
        R1 = C; G1 = X; B1 = 0;
        break;
    case 1:
        R1 = X; G1 = C; B1 = 0;
        break;
    case 2:
        R1 = 0; G1 = C; B1 = X;
        break;
    case 3:
        R1 = 0; G1 = X; B1 = C;
        break;
    case 4:
        R1 = X; G1 = 0; B1 = C;
        break;
    case 5:
        R1 = C; G1 = 0; B1 = X;
        break;
    default:
        R1 = 0; G1 = 0; B1 = 0;
        break;
    }

    TFloat m = V - C;
    int r = (int)((R1 + m) * 255);
    int g = (int)((G1 + m) * 255);
    int b = (int)((B1 + m) * 255);
    return (0 << 24) | (r << 16) | (g << 8) | (b << 0);
}

__global__ void juliaset(int *buffer, int width, int height,
    int maxIteration, TFloat cx, TFloat cy, TFloat minX, TFloat maxX, TFloat minY, TFloat maxY)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    int idx = y * width + x;
    TFloat fx = minX + (maxX - minX) * x / width;
    TFloat fy = minY + (maxY - minY) * y / height;

    TFloat zx = fx;
    TFloat zy = fy;
    TFloat iteration = 0;
    for (; iteration < maxIteration && zx * zx + zy * zy < 4; iteration += 1)
    {
        TFloat newZx = zx * zx - zy * zy + cx;
        zy = 2 * zx * zy + cy;
        zx = newZx;
    }

    if (iteration < maxIteration)
    {
        TFloat logZn = log(zx * zx + zy * zy) / 2;
        TFloat nu = log(logZn / log(2.0)) / log(2.0);
        iteration = iteration + 1 - nu;
    }

    TFloat value = iteration / maxIteration;
    buffer[idx] = HSV2RGB(value, 1 - value * value, sqrt(value));
}

__global__ void mandelbrot(int *buffer, int width, int height,
    int maxIteration, TFloat minX, TFloat maxX, TFloat minY, TFloat maxY)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    int idx = y * width + x;
    TFloat fx = minX + (maxX - minX) * x / width;
    TFloat fy = minY + (maxY - minY) * y / height;

    TFloat zx = 0;
    TFloat zy = 0;
    TFloat iteration = 0;
    for (; iteration < maxIteration && zx * zx + zy * zy < 4; iteration += 1)
    {
        TFloat newZx = zx * zx - zy * zy + fx;
        zy = 2 * zx * zy + fy;
        zx = newZx;
    }

    if (iteration < maxIteration)
    {
        TFloat logZn = log(zx * zx + zy * zy) / 2;
        TFloat nu = log(logZn / log(2.0)) / log(2.0);
        iteration = iteration + 1 - nu;
    }

    TFloat value = iteration / maxIteration;
    buffer[idx] = HSV2RGB(value, 1 - value * value, sqrt(value));
}

class CUDAPinnedPtr
{
public:
    CUDAPinnedPtr(void *hostPtr, int size)
        : mHostPtr(hostPtr), mDevicePtr(nullptr)
    {
        cudaCheck(hipHostRegister(mHostPtr, size, hipHostRegisterMapped));
        cudaCheck(hipHostGetDevicePointer(&mDevicePtr, mHostPtr, 0));
    }

    ~CUDAPinnedPtr()
    {
        cudaCheck(hipHostUnregister(mHostPtr));
    }

    void *HostPtr() const
    {
        return mHostPtr;
    }

    void *DevicePtr() const
    {
        return mDevicePtr;
    }

private:
    void *mHostPtr;
    void *mDevicePtr;
};

class CUDAFractalRenderer : public IFractalRenderer
{
public:
    CUDAFractalRenderer(int width, int height)
    {
        for (int i = 0; i < 2; ++i)
        {
            mStreams.push_back(0);
            cudaCheck(hipStreamCreate(&mStreams.back()));
        }
    }

    ~CUDAFractalRenderer()
    {
        for (auto stream : mStreams)
        {
            hipStreamDestroy(stream);
        }
        mStreams.clear();
    }

    virtual void ResetBuffer(int width, int height)
    {   
    }

    virtual void RenderMandelbrot(
        int *buffer, int width, int height, int maxIteration,
        TFloat minX, TFloat maxX, TFloat minY, TFloat maxY)
    {
        auto deviceBuffer = (int*)GetCachedDevicePtr(buffer, width, height);

        int tileHeight = height / (int)mStreams.size();
        TFloat tileY = (maxY - minY) / (int)mStreams.size();
        for (int i = 0; i < (int)mStreams.size(); ++i)
        {
            dim3 dimBlock(kBlockSize, kBlockSize);
            dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (tileHeight + dimBlock.y - 1) / dimBlock.y);

            TFloat tileMinY = minY + tileY * i;
            TFloat tileMaxY = tileMinY + tileY;

            mandelbrot << <dimGrid, dimBlock, 0, mStreams[i] >> >(deviceBuffer + i * tileHeight * width, width, tileHeight, maxIteration, minX, maxX, tileMinY, tileMaxY);
        }

        for (auto stream : mStreams)
        {
            cudaCheck(hipStreamSynchronize(stream));
        }
    }

    virtual void RenderJuliaSet(
        int *buffer, int width, int height, int maxIteration, TFloat cx, TFloat cy,
        TFloat minX, TFloat maxX, TFloat minY, TFloat maxY)
    {
        auto deviceBuffer = (int*)GetCachedDevicePtr(buffer, width, height);

        int tileHeight = height / (int)mStreams.size();
        TFloat tileY = (maxY - minY) / (int)mStreams.size();
        for (int i = 0; i < (int)mStreams.size(); ++i)
        {
            dim3 dimBlock(kBlockSize, kBlockSize);
            dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (tileHeight + dimBlock.y - 1) / dimBlock.y);

            TFloat tileMinY = minY + tileY * i;
            TFloat tileMaxY = tileMinY + tileY;

            juliaset << <dimGrid, dimBlock, 0, mStreams[i] >> >(deviceBuffer + i * tileHeight * width, width, tileHeight, maxIteration, cx, cy, minX, maxX, tileMinY, tileMaxY);
        }

        for (auto stream : mStreams)
        {
            cudaCheck(hipStreamSynchronize(stream));
        }
    }

private:
    void *GetCachedDevicePtr(int *buffer, int width, int height)
    {
        int index = -1;
        for (int i = 0; i < (int)mPinnedPtrs.size(); ++i)
        {
            if (mPinnedPtrs[i]->HostPtr() == buffer) index = i;
        }
        if (index == -1)
        {
            if (mPinnedPtrs.size() == 4) mPinnedPtrs.pop_back();
            mPinnedPtrs.insert(mPinnedPtrs.begin(), std::make_unique<CUDAPinnedPtr>(buffer, width * height * sizeof(*buffer)));
            index = 0;
        }
        return mPinnedPtrs[index]->DevicePtr();
    }

private:
    int mWidth, mHeight;
    std::vector<hipStream_t> mStreams;
    std::vector<std::unique_ptr<CUDAPinnedPtr>> mPinnedPtrs;
};

IFractalRenderer* CreateCUDAFractalRenderer(int width, int height)
{
    return new CUDAFractalRenderer(width, height);
}