#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#include <vector>
#include <memory>

#include "hip/hip_runtime.h"
#include ""
#include "FractalRenderer.h"

#if USE_DOUBLE
#define ufmod   fmod
#else
#define ufmod   fmodf
#endif

static const int kBlockSize = 32;

#define cudaCheck(call) { cudaCheckError((call), __FILE__, __LINE__); }
inline void cudaCheckError(hipError_t error, const char *file, int line, bool abort = true)
{
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA error: %s %s %d\n", hipGetErrorString(error), file, line);
        if (abort) exit(error);
    }
}

__device__ static int HSV2RGB(TFloat H, TFloat S, TFloat V)
{
    TFloat C = V * S;
    TFloat H1 = H * 6;
    TFloat X = C * (1 - fabs(ufmod(H1, 2.0) - 1));
    TFloat R1, G1, B1;
    switch ((int)floor(H1))
    {
    case 0:
        R1 = C; G1 = X; B1 = 0;
        break;
    case 1:
        R1 = X; G1 = C; B1 = 0;
        break;
    case 2:
        R1 = 0; G1 = C; B1 = X;
        break;
    case 3:
        R1 = 0; G1 = X; B1 = C;
        break;
    case 4:
        R1 = X; G1 = 0; B1 = C;
        break;
    case 5:
        R1 = C; G1 = 0; B1 = X;
        break;
    default:
        R1 = 0; G1 = 0; B1 = 0;
        break;
    }

    TFloat m = V - C;
    int r = (int)((R1 + m) * 255);
    int g = (int)((G1 + m) * 255);
    int b = (int)((B1 + m) * 255);
    return (0 << 24) | (r << 16) | (g << 8) | (b << 0);
}

__global__ void juliaset(int *buffer, int width, int height,
    int maxIteration, TFloat cx, TFloat cy, TFloat minX, TFloat maxX, TFloat minY, TFloat maxY)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    int idx = y * width + x;
    TFloat fx = minX + (maxX - minX) * x / width;
    TFloat fy = minY + (maxY - minY) * y / height;

    TFloat zx = fx;
    TFloat zy = fy;
    TFloat iteration = 0;
    for (; iteration < maxIteration && zx * zx + zy * zy < 4; iteration += 1)
    {
        TFloat newZx = zx * zx - zy * zy + cx;
        zy = 2 * zx * zy + cy;
        zx = newZx;
    }

    if (iteration < maxIteration)
    {
        TFloat logZn = log(zx * zx + zy * zy) / 2;
        TFloat nu = log(logZn / log(2.0)) / log(2.0);
        iteration = iteration + 1 - nu;
    }

    TFloat value = iteration / maxIteration;
    buffer[idx] = HSV2RGB(value, 1 - value * value, sqrt(value));
}

__global__ void mandelbrot(int *buffer, int width, int height,
    int maxIteration, TFloat minX, TFloat maxX, TFloat minY, TFloat maxY)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    int idx = y * width + x;
    TFloat fx = minX + (maxX - minX) * x / width;
    TFloat fy = minY + (maxY - minY) * y / height;

    TFloat zx = 0;
    TFloat zy = 0;
    TFloat iteration = 0;
    for (; iteration < maxIteration && zx * zx + zy * zy < 4; iteration += 1)
    {
        TFloat newZx = zx * zx - zy * zy + fx;
        zy = 2 * zx * zy + fy;
        zx = newZx;
    }

    if (iteration < maxIteration)
    {
        TFloat logZn = log(zx * zx + zy * zy) / 2;
        TFloat nu = log(logZn / log(2.0)) / log(2.0);
        iteration = iteration + 1 - nu;
    }

    TFloat value = iteration / maxIteration;
    buffer[idx] = HSV2RGB(value, 1 - value * value, sqrt(value));
}

class CUDAPinnedPtr
{
public:
    CUDAPinnedPtr(void *hostPtr, int size)
        : mHostPtr(hostPtr), mDevicePtr(nullptr)
    {
        cudaCheck(hipHostRegister(mHostPtr, size, hipHostRegisterMapped));
        cudaCheck(hipHostGetDevicePointer(&mDevicePtr, mHostPtr, 0));
    }

    ~CUDAPinnedPtr()
    {
        cudaCheck(hipHostUnregister(mHostPtr));
    }

    void *HostPtr() const
    {
        return mHostPtr;
    }

    void *DevicePtr() const
    {
        return mDevicePtr;
    }

private:
    void *mHostPtr;
    void *mDevicePtr;
};

class CUDAFractalRenderer : public IFractalRenderer
{
public:
    CUDAFractalRenderer(int width, int height)
    {
        
    }

    ~CUDAFractalRenderer()
    {
    }

    virtual void ResetBuffer(int width, int height)
    {   
    }

    virtual void RenderMandelbrot(
        int *buffer, int width, int height, int maxIteration,
        TFloat minX, TFloat maxX, TFloat minY, TFloat maxY)
    {
        dim3 dimBlock(kBlockSize, kBlockSize);
        dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);
        
        auto deviceBuffer = (int*)GetCachedDevicePtr(buffer, width, height);
        mandelbrot << <dimGrid, dimBlock >> >(deviceBuffer, width, height, maxIteration, minX, maxX, minY, maxY);
    }

    virtual void RenderJuliaSet(
        int *buffer, int width, int height, int maxIteration, TFloat cx, TFloat cy,
        TFloat minX, TFloat maxX, TFloat minY, TFloat maxY)
    {
        dim3 dimBlock(kBlockSize, kBlockSize);
        dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);

        auto deviceBuffer = (int*)GetCachedDevicePtr(buffer, width, height);
        juliaset << <dimGrid, dimBlock >> >(deviceBuffer, width, height, maxIteration, cx, cy, minX, maxX, minY, maxY);
    }

private:
    void *GetCachedDevicePtr(int *buffer, int width, int height)
    {
        int index = -1;
        for (int i = 0; i < (int)mPinnedPtrs.size(); ++i)
        {
            if (mPinnedPtrs[i]->HostPtr() == buffer) index = i;
        }
        if (index == -1)
        {
            if (mPinnedPtrs.size() == 4) mPinnedPtrs.pop_back();
            mPinnedPtrs.insert(mPinnedPtrs.begin(), std::make_unique<CUDAPinnedPtr>(buffer, width * height * sizeof(*buffer)));
            index = 0;
        }
        return mPinnedPtrs[index]->DevicePtr();
    }

private:
    int mWidth, mHeight;
    std::vector<std::unique_ptr<CUDAPinnedPtr>> mPinnedPtrs;
};

IFractalRenderer* CreateCUDAFractalRenderer(int width, int height)
{
    return new CUDAFractalRenderer(width, height);
}